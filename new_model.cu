//////////////////////////////////////////New code////////////////////////////////////


#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include<set>
#include<fstream>
#include <chrono>
#include<stdio.h>


using namespace std::chrono;
using std::ifstream;
ifstream indata ; 

#define ll long long int 


using namespace std;
/*----------------------------------------------------------------------------*/

__device__ bool check_edge(ll* edges  , ll left ,ll right , ll ff , ll ss )
{
 
    while(left<=right)
    {
        ll center= (right-left)/2 + left ; 
   //  printf("%d %d %d %d \n" , edges[center*2] , edges[center*2+1] , ff , ss  ) ; 
        if(edges[center*2]== ff && edges[center*2+1] == ss)
        return 1 ;  //Found 

        if(edges[center*2] > ff )
        {
            right = center -1 ; 
        }else if(edges[center*2] < ff )
        left = center+1 ; 
        else if (edges[center*2+1] < ss)
        left = center +1 ; 
        else 
        right = center -1 ; 
     }
    return 0 ; 
}

/*----------------------------------------------------------------------------*/

__device__ int match_value(ll* last_cliq ,ll center ,ll index ,ll cli )
{
    for(ll i =0 ;i <cli-1 ;i++)
    {
        if(last_cliq[center*cli+i] > last_cliq[index*cli+i+1])
        return 1 ;
        else if (last_cliq[center*cli+i] < last_cliq[index*cli+i+1])
        return -1 ; 
    }
    return 0 ; 
}
/*----------------------------------------------------------------------------*/
__device__ ll find_starting_point(ll* last_cliq ,ll left ,  ll right  ,ll cli , ll index )
{
    ll ans = -1 ; 
    while(left <= right )
    {
        ll center = (right-left)/2 +left ;
        if(match_value(last_cliq , center , index , cli ) == 0 )
        {
            ans = center ; 
            right =center -1 ; 
        } else if (match_value(last_cliq , center , index , cli ) == -1 )
        {   
            
            
            //center is less than index 
            left = center +1 ; 
        }else 
          right = center -1 ; 
    }
 return  ans ; 
} 
/*----------------------------------------------------------------------------*/
__global__ void count_next_cli (ll* edges ,ll* last_cliq ,ll n ,ll m ,ll current_cli_size ,ll*  start_3_cli , ll cli  )
{
    
      ll index = blockDim.x * blockIdx.x + threadIdx.x ;
      if(index < current_cli_size)
      {   
           
           start_3_cli[index] = 0 ; //Preset value 
          ll starting_point = find_starting_point(last_cliq ,0 ,  current_cli_size-1 , cli , index ) ;
          if(starting_point == -1 )
          return ;    
          for( ; starting_point < current_cli_size && match_value(last_cliq , starting_point , index , cli )==0; starting_point++)
          {   bool added = 1 ; 

              //printf("%d\n" , starting_point ) ; 
              //So we need to see that is the last element connected to all 
              for(ll j = 0 ; j <cli ; j++ )
              {
                  if(check_edge(edges , 0 , m -1 , last_cliq[starting_point*cli+cli-1] , last_cliq[index*cli+j]  )==0)
                  {
                      added = 0 ;
                  }
              }
           start_3_cli[index] += added  ; 
                  
          }
      }
}


__global__ void make_next_cli (ll* edges ,ll* last_cliq ,ll n ,ll m ,ll current_cli_size ,ll*  places , ll cli , ll* update  )
{
      ll index = blockDim.x * blockIdx.x + threadIdx.x ;
      if(index < current_cli_size)
      {   
          ll starting_point = find_starting_point(last_cliq ,0 ,  current_cli_size-1 , cli , index ) ;
          if(starting_point == -1 )
          return ; 
          ll writing_ptr = places[index] ; 
          for( ; starting_point < current_cli_size && match_value(last_cliq , starting_point , index , cli )==0; starting_point++)
          {   bool added = 1 ; 
            
              //So we need to see that is the last element connected to all 
              for(ll j = 0 ; j <cli ; j++ )
              {
                  if(check_edge(edges , 0 , m -1 , last_cliq[starting_point*cli+cli-1] , last_cliq[index*cli+j]  )==0)
                  added = 0 ;

                    
              }
           if(added)
            {
                for(ll i =0 ;i < cli  ; i++)
                {
                    update[writing_ptr*(cli+1)+i] = last_cliq[index*cli+i] ; 
                 
                }
                    update[writing_ptr*(cli+1)+cli] = last_cliq[starting_point*cli+cli-1] ; //I want the last element 
                  writing_ptr++ ; 
            }
                  

           
          }
      }
}





void  make_two_cli(ll m  , vector<ll>& host_edges , vector<ll>& host_two_cli  )
{ 
    set<pair<ll,ll>> tem ;  
    for(ll i =0;i<m;i++)
    {
        ll a,b ; indata>>a>>b ; 
        tem.insert({a,b})  ;     
        tem.insert({b,a})  ;    //Considering the undirected edges 
    
    }
    for(auto i : tem)
    {
        /* Directed 
            if(i.first < i.second && (tem.find({i.second , i.first }) != tem.end()) )
                {host_two_cli.push_back(i.first) ;  host_two_cli.push_back(i.second) ;} 
            host_edges.push_back(i.first);host_edges.push_back(i.second); //It will be automatically sorted 
     
        */
        
        /*Undirected*/
        
        host_edges.push_back(i.first);host_edges.push_back(i.second); //It will be automatically sorted 
        
      if(i.first < i.second )
         {host_two_cli.push_back(i.first) ;  host_two_cli.push_back(i.second) ; }
       
     
    }
 //So I have updated  the value of both 
}

ll find_kcliq(ll k)
{
    ll  m ;indata>>m ;
    ll n = 1 ;
    /// take the input 
    vector<ll> host_edges; 
    vector<ll> host_cli;
    make_two_cli(m , host_edges , host_cli) ;
    ll current_cli_size = host_cli.size()/2 ; //Num of cliques  
    for(auto i : host_cli)
      n = max(i,n) ; 
    if(k==1)
    return n ; 
    if(k==2)
    return current_cli_size ; 
    n++ ; 
    m*=2 ; //only for undirected graph 
    ll cli = 2 ;
 
    //Upload the edges data on the 

       ll* device_edges   ; 
       hipMalloc( &device_edges , sizeof(ll)*m*2) ;
       hipMemcpy(device_edges ,host_edges.data()  , sizeof(ll)*(m*2) , hipMemcpyHostToDevice )  ;   
       ll* last_cliq ;
       hipMalloc( &last_cliq , sizeof(ll)*current_cli_size*cli) ;
       hipMemcpy(last_cliq, host_cli.data()  , sizeof(ll)*current_cli_size*cli , hipMemcpyHostToDevice )  ;
      //  cout << current_cli_size <<"\n" ; 
        while(cli<k)
        {
            //Find the number of cliques for cli+1 
            ll* device_number_cli ; 
            hipMalloc( &device_number_cli , sizeof(ll)*current_cli_size) ;
            int threadsPerBlock = 100 ;
            ll blocksPerGrid = ( (current_cli_size)+ threadsPerBlock - 1) / threadsPerBlock;
            count_next_cli<<<blocksPerGrid, threadsPerBlock>>> (device_edges , last_cliq ,n , m , current_cli_size ,  device_number_cli , cli );
             
            ll new_next_cliq = 0 ; 
            vector<ll> number_cli(current_cli_size);
            hipMemcpy(number_cli.data() , device_number_cli, sizeof(ll)*(current_cli_size) , hipMemcpyDeviceToHost);
            for(ll i=0;i<current_cli_size ; i++)
            {
                ll tem = number_cli[i] ; 
                number_cli[i] = new_next_cliq ; //It becomes the starting point  
                new_next_cliq +=  tem ; 
             
            }
         
            if(cli+1 < k)
            {
                //So I would require to add new value 
                ll*  dummy_last_cli ; 
                hipMalloc(&dummy_last_cli , sizeof(ll)*new_next_cliq*(cli+1)) ; //So this will be the length 
                //We would had to copy it back 
                hipMemcpy(  device_number_cli,number_cli.data() ,  sizeof(ll)*(current_cli_size) , hipMemcpyHostToDevice);
                make_next_cli<<<blocksPerGrid, threadsPerBlock>>> (device_edges , last_cliq ,n , m , current_cli_size ,  device_number_cli , cli , dummy_last_cli );

                hipFree(last_cliq) ; 
                last_cliq =  dummy_last_cli ;        
                vector<ll> ads (new_next_cliq*(cli+1 )) ; 

                hipMemcpy(ads.data() ,last_cliq, sizeof(ll)*(new_next_cliq*(cli+1 )) , hipMemcpyDeviceToHost);


                //We need no memory copy 
            }

            //We need to update the values now 
            cli++ ; 
            current_cli_size = new_next_cliq ; 



        }   
 
      hipFree(last_cliq) ; 
      hipFree(device_edges) ; 
      return current_cli_size ; 
       
 
}
int main()
{
    
    ll k  ;
    string file_name   ;
  
    cin>>file_name >> k ; 
//    cout<<file_name <<" " << k << "\n" ; 
    indata.open(file_name); // opens the file
    if(!indata) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
    }
  milliseconds ms = duration_cast< milliseconds >(
    system_clock::now().time_since_epoch()
);
 
    ll ms1 =std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    ll ans  =  find_kcliq(k) ;
    ll ms2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    
    cout<< ans  <<"\nExecution Time: "<< ms2 - ms1 <<"ms\n" ;

}
