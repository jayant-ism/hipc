
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include<algorithm>
#include <ctime>
#include<map>

using namespace std;
#define ll long long int 



#include <fstream>
using std::ifstream;

     ifstream indata; // indata is like cin



//The vector file required 

template<typename T>
class LocalVector
{
private:
    T* m_begin;
    T* m_end;

    size_t capacity;
    __device__ void expand() {
        capacity *= 2;
        size_t tempLength = (m_end - m_begin);
        T* tempBegin = new T[capacity];

        memcpy(tempBegin, m_begin, tempLength * sizeof(T));
        delete[] m_begin;
        m_begin = tempBegin;
        m_end = m_begin + tempLength;
        length = static_cast<size_t>(m_end - m_begin);
    }
public:
    size_t length;
 
    __device__  explicit LocalVector() : length(0), capacity(16) {
        m_begin = new T[capacity];
        m_end = m_begin;
    }
    __device__ T& operator[] (unsigned int index) {
        return *(m_begin + index);//*(begin+index)
    }
    __device__ T* begin() {
        return m_begin;
    }
    __device__ T* end() {
        return m_end;
    }
    __device__ ~LocalVector()
    {
        delete[] m_begin;
        m_begin = nullptr;
    }

    __device__ void add(T t) {

        if ((m_end - m_begin) >= capacity) {
            expand();
        }

        new (m_end) T(t);
        m_end++;
        length++;
    }
    __device__ T pop() {
        T endElement = (*m_end);
        delete m_end;
        m_end--;
        return endElement;
    }

    __device__ size_t getSize() {
        return length;
    }
};



/////////////////////////////////



//Sorting algo
void sort( ll *edges , ll m )
{
  vector<pair<ll,ll>> arr ;
  for(ll i =0;i<m;i++)
  arr.push_back({edges[i*2] , edges[i*2+1]}) ; 
  sort(arr.begin() , arr.end()) ; 
 for(ll i =0;i<m;i++)
 {
     edges[i*2] = arr[i].first ; 
     edges[i*2+1] = arr[i].second ; 
 }

}

///////////////////////////////

__device__ bool check_edge(ll* edges  , ll left ,ll right , ll ff , ll ss )
{
 
    while(left<=right)
    {
        ll center= (right-left)/2 + left ; 
   //  printf("%d %d %d %d \n" , edges[center*2] , edges[center*2+1] , ff , ss  ) ; 
        if(edges[center*2]== ff && edges[center*2+1] == ss)
        return 1 ;  //Found 

        if(edges[center*2] > ff )
        {
            right = center -1 ; 
        }else if(edges[center*2] < ff )
        left = center+1 ; 
        else if (edges[center*2+1] < ss)
        left = center +1 ; 
        else 
        right = center -1 ; 
     }
    return 0 ; 
}

__device__ ll first_node_greater_than_n(ll* edges , ll left ,ll right , ll n )
{   ll ans = -1  ; 
    while(left<= right )
    {
        ll center = (right - left)/2 + left ; 

        if(edges[center*2] > n )
        {
            ans =  center ; 
            right = center-1; 
        }else 
          left = center +1 ; 
    }
 return ans ; 

}


//use two clique beacause its distinct 
__device__ ll start_ele(ll * two_cliq , ll m , ll ele )
{
    ll left = 0 , right = m-1 ; 
    ll ans = -1 ; 

 while(left <= right )
 {
    ll center = (right - left ) /2 + left ; 
  if(two_cliq[center*2] == ele  ) 
  ans = center ; 

  if(two_cliq[center*2] >=  ele )
  right = center -1 ; 
  else 
  left = center +1 ;  
 
 }
 return ans ; 

}




//---------------------------------------------------------------------------------------------------------
//For k<=4 

__device__ ll sol(ll*edges , LocalVector<ll> &arr  , ll k , ll now ,  ll n ,ll m , ll* two_cliques )
{
 
    if(k==now)
    return 1 ; 
    ll ans =0 ; 

    if(k-2 >= now )
    {   
        ll starting_index = arr[now-1] ; 
        starting_index  = first_node_greater_than_n(two_cliques , 0, (m-1)/2,  starting_index) ; 
        if(starting_index == -1 )
        return 0 ; 
  //   printf("%d\n"  , starting_index ) ;

        for( ; starting_index < m/2 ;starting_index++)
        {
            bool to_add = 1 ;
         for(ll i = 0 ; i < now ;i++)
         {
             if( !( check_edge(edges , 0 , m-1 , arr[i] , two_cliques[starting_index*2]   ) && check_edge(edges , 0 , m-1 , arr[i] , two_cliques[starting_index*2+1]   )  || arr[i]  >= two_cliques[starting_index*2]) )
            {
                to_add = 0 ;
                break ; 
            }
         }
          if(to_add == 1 )
          {   
              arr[now] = two_cliques[starting_index*2] ;

              arr[now+1] = two_cliques[starting_index*2+1] ; 
              
              ans +=  sol(edges , arr , k , now+2 , n , m , two_cliques) ; 
              
          }
        }
       
    }else 
    {
   
    ll starting_element =  start_ele(two_cliques , m/2 , arr[now-1]  ) ; 
  //  printf("%d\n", starting_element ) ; 
      
        if(starting_element == -1 )
        return 0 ; 
     ll sd = arr[3] - arr[2] ; 
   //     printf("%d\n" , sd ) ;
      for(    ; starting_element < m/2 && two_cliques[starting_element*2] == arr[now-1] ; starting_element++  )
      {
           ll i = two_cliques[starting_element*2+1] ;
    
           bool added = 1 ; 
          for(ll j = 0 ; j < now ; j++ )
          {
              if(check_edge(edges , 0 , m-1,  i , arr[j]) == 0 )
              {
                  added = 0 ;
                  break ;  
              }
          }
       //So we can work with i 
        if(added)
        {
       //     printf("%d %d %d %d %d %d %d %d %d %d\n" , arr[0] , arr[0] , arr[1] , arr[1] , arr[2] , arr[2] ,arr[3] ,arr[3] , arr[4] ,arr[4]) ;
            arr[now] = i ; 
            ans +=  1 ; 
        }
       } 
    }
 return ans ; 
}

__global__ void doi( ll* edges , ll * updates , ll *two_cliques , ll n , ll m , ll k )
{
    
    ll index = blockDim.x * blockIdx.x + threadIdx.x ;
    if(index <  m/2)
    {
        
     
        if(k==2)
        {
            updates[index] = 1 ; 
            return ; 
        }
      
         //Now we will find the 3-cliques 

        ll current_size = 0 ;

     
              //  printf("%d\n", two_cliques[2*index+1]+1)  ;
                   LocalVector<ll> dum  ; 
                    dum.add( two_cliques[index*2]  ) ;
                    dum.add( two_cliques[index*2+1]  ) ;
                  for(ll i =0;i<k-2 ; i++ )
                  dum.add(0) ;
            current_size += sol(edges , dum , k , 2 , n , m , two_cliques ) ; 
         

     
         updates[index]  = current_size  ; 
         return ; 
    
    }    
}




//--------------------------------------------------------------------------------

__global__ void count_three_cliques (ll* edges ,ll*  two_cliques,ll    n ,ll  m , ll* start_3_cli    ) 
      {
          ll index = blockDim.x * blockIdx.x + threadIdx.x ;
       if(index<m/2)
       {
       
          ll ans = 0 ;    
          ll starting_element =  start_ele(two_cliques , m/2 , two_cliques[index*2+1]  ) ; 

          if(starting_element == -1 )
          return  ; 
          for(    ; starting_element < m/2 && two_cliques[starting_element*2] ==  two_cliques[index*2+1] ; starting_element++  )
          {
           ll i = two_cliques[starting_element*2+1] ;
    
           bool added = 1 ; 
          for(ll j = 0 ; j < 2; j++ )
          {
              if(check_edge(edges , 0 , m-1,  i , two_cliques[index*2+j] ) == 0 )
              {
                  added = 0 ;
                  break ;  
              }
          }
       //So we can work with i 
        if(added)
        {
           
            ans +=  1 ; 
        }
       }
        
          start_3_cli[index] =ans  ; 

      }
  }

      
//I have counted the number of 3 cliques , lets construct them 



__global__ void make_three_cliques (ll* edges ,ll*  two_cliques,ll    n ,ll  m , ll* start_3_cli , ll* three_cliques   ) 
      {
          
          ll index = blockDim.x * blockIdx.x + threadIdx.x ;
       if(index<m/2)
       {

           
           ll start_index = start_3_cli[index]  ; 

          ll ans = 0 ;    
          ll starting_element =  start_ele(two_cliques , m/2 , two_cliques[index*2+1]  ) ; 

          if(starting_element == -1 )
          return  ; 
          for(    ; starting_element < m/2 && two_cliques[starting_element*2] ==  two_cliques[index*2+1] ; starting_element++  )
          {
           ll i = two_cliques[starting_element*2+1] ;
    
           bool added = 1 ; 
          for(ll j = 0 ; j < 2; j++ )
          {
              if(check_edge(edges , 0 , m-1,  i , two_cliques[index*2+j] ) == 0 )
              {
                  added = 0 ;
                  break ;  
              }
          }
       //So we can work with i 
        if(added)
        {
                    three_cliques[start_index*3] =  two_cliques[index*2] ; 
                    three_cliques[start_index*3+1] =  two_cliques[index*2+1] ; 
                    three_cliques[start_index*3+2] =  i ; 
                    start_index++ ;  
            ans +=  1 ; 
        }
       }
        

        
        
        }

      }
//-----------------------------------------------------------------
//I will have to find the solution



__device__ ll find_starting_three(ll* three_cliques , ll left , ll right  ,  ll starting_element )
{
    ll ans = -1 ; 
 while(left<=right)
 {
     ll center = (right - left)/2 + left ; 
  if(three_cliques[center*3] == starting_element )
  {
      ans = center ; 
  }
  if(three_cliques [center*3] >= starting_element )
  {
      right = center -1 ; 
  } else 
  left = center +1 ; 
 }
 return ans ; 

}
__global__ void solution_greater (ll* edges ,ll*  two_cliques, ll* three_cliques,ll  n ,ll m ,ll k  ,ll three_size ,ll* update)
{
       ll index = blockDim.x * blockIdx.x + threadIdx.x ;

     if(index < three_size)
     {  
    //     printf("%d\n", index) ; 
       update[index]= 0 ; 
          if(k==4)
          {
              
            
            ll ans = 0 ;    
            ll starting_element =  start_ele(two_cliques , m/2 , three_cliques[index*3+2]  ) ; 

            if(starting_element == -1 )
            return  ; 
           
            for(    ; starting_element < m/2 && two_cliques[starting_element*2] ==  three_cliques[index*3+2] ; starting_element++  )
            {
             
             
              int counted = 1 ; 
                for(int j =0 ; j < 2 ; j++)
                {
                    for(int k = 0; k <3 ; k++)
                    {
                        //three clique .. two_clique 
                     if(j!=0 && k!= 2 )
                        if(check_edge(edges,0,m-1,two_cliques[starting_element*2+j ] , three_cliques[index*3+k])== 0   )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
             update[index] += counted ; 




            }
        



          }else if (k==5)
         {
             
             ll starting_element = find_starting_three(three_cliques , 0 , three_size -1 ,  three_cliques[index*3+2]) ; 
             if(starting_element == -1 )
             {
                 return ; 
             }
          
          for( ; starting_element< three_size && three_cliques[starting_element*3] ==  three_cliques[index*3+2] ; starting_element++ )
          {
                int counted = 1 ; 
                for(int j =0 ; j < 3 ; j++)
                {
                    for(int k = 0; k <3 ; k++)
                    {
                        //three clique .. two_clique 
                     if(j!=0 && k!=2  )
                        if(check_edge(edges,0,m-1, three_cliques[starting_element*3+j ] , three_cliques[index*3+k])== 0    )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
             update[index] += counted ; 

          }



         }else if(k==6)
       {
           
            
             
             ll starting_element = find_starting_three(three_cliques , 0 , three_size -1 ,  three_cliques[index*3+2]) ; 
             if(starting_element == -1 )
             {
                 return ; 
             }
          
          for( ; starting_element< three_size && three_cliques[starting_element*3] ==  three_cliques[index*3+2] ; starting_element++ )
          {
                int counted = 1 ; 
                for(int j =0 ; j < 3 ; j++)
                {
                    for(int k = 0; k <3 ; k++)
                    {
                        //three clique .. two_clique 
                     if(j!=0 && k!=2  )
                        if(check_edge(edges,0,m-1, three_cliques[starting_element*3+j ] , three_cliques[index*3+k])== 0    )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
           if(counted )
           {


                   LocalVector<ll> dum  ; 
                    dum.add( three_cliques[index*3]  ) ;
                    dum.add( three_cliques[index*3+1]  ) ;
                    dum.add( three_cliques[index*3+2]  ) ;
                    dum.add( three_cliques[starting_element*3+1]  ) ;
                    dum.add( three_cliques[starting_element*3+2]  ) ;
                    dum.add(0) ; 

              update[index] +=  sol(edges , dum , k , 5 , n , m , two_cliques ) ; 
         
 
  
           }
             
          }


       }else if(k==7)
       {
           

  
             ll starting_element = find_starting_three(three_cliques , 0 , three_size -1 ,  three_cliques[index*3+2]) ; 
             if(starting_element == -1 )
             {
                 return ; 
             }
          
          for( ; starting_element< three_size && three_cliques[starting_element*3] ==  three_cliques[index*3+2] ; starting_element++ )
          {
                int counted = 1 ; 
                for(int j =0 ; j < 3 ; j++)
                {
                    for(int k = 0; k <3 ; k++)
                    {
                        //three clique .. two_clique 
                     if(j!=0 && k!=2  )
                        if(check_edge(edges,0,m-1, three_cliques[starting_element*3+j ] , three_cliques[index*3+k])== 0    )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
           if(counted == 1 )
           {
               
                   LocalVector<ll> dum  ; 
                    dum.add( three_cliques[index*3]  ) ;
                    dum.add( three_cliques[index*3+1]  ) ;
                    dum.add( three_cliques[index*3+2]  ) ;
                    dum.add( three_cliques[starting_element*3+1]  ) ;
                    dum.add( three_cliques[starting_element*3+2]  ) ;
  
            
        
                  ll second_starting = find_starting_three(three_cliques , 0 , three_size -1 , dum[4] )  ;
                  if(second_starting == -1 )
                    continue ; 
                  for( ; second_starting < three_size && three_cliques[second_starting*3] == dum[4] ; second_starting++ )
                  {
                    bool okd = 1 ; 
                   for(ll gi = 0 ; gi <5 ;gi++)
                   {
                       for(ll gj  = 1  ; gj < 3 ; gj++ )

                       {
                           if(check_edge(edges , 0 , m-1, dum[gi] ,  three_cliques[second_starting*3+gj] )==0 )
                           okd = 0 ; 
                       }
                   }
                    update[index] += okd  ; 
                       
                  }
          }
          
          }



           
       }
      
     }
      
}


//-------------------------------------------------------------------------------------------------------------
//Values for 6 && 7 



__global__ void make_four_clique (ll* edges ,ll*  two_cliques, ll* three_cliques,ll  n ,ll m ,ll k  ,ll three_size ,ll* four_clique_sizes , ll* four_cliques)
{

     ll index = blockDim.x * blockIdx.x + threadIdx.x ;

     if(index < three_size)
     {  
          
              
            ll start_writing = four_clique_sizes[index] ; 
              
            ll ans = 0 ;    
            ll starting_element =  start_ele(two_cliques , m/2 , three_cliques[index*3+2]  ) ; 

            if(starting_element == -1 )
            return  ; 
           
            for(    ; starting_element < m/2 && two_cliques[starting_element*2] ==  three_cliques[index*3+2] ; starting_element++  )
            {
             
             
              int counted = 1 ; 
                for(int j =0 ; j < 2 ; j++)
                {
                    for(int k = 0; k <3 ; k++)
                    {
                        //three clique .. two_clique 
                     if(j!=0 && k!= 2 )
                        if(check_edge(edges,0,m-1,two_cliques[starting_element*2+j ] , three_cliques[index*3+k])== 0   )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
             if(counted)
             {
                 four_cliques[start_writing*4 ] = three_cliques[index*3] ; 
                 four_cliques[start_writing*4 +1] = three_cliques[index*3+1] ; 
                 four_cliques[start_writing*4 +2 ] = three_cliques[index*3+2] ; 
                 four_cliques[start_writing*4 +3 ] = two_cliques[starting_element*2+1] ; 
                start_writing++ ; 
             }
             

            }

      }


}

//------------------------------------------------------------------------

//Make  for 6 & 7 ------------------------------------------------------


__device__ ll find_starting_four(ll* four_cliques , ll left  ,ll right  ,  ll search )
{
    ll ans = -1 ; 
     while(left <= right )
     {
        ll center = (right - left)/2 + left ; 
        if(four_cliques[center*4] ==  search )
        ans = center ; 
        if(four_cliques[center*4] >= search)
        {
            right = center - 1; 
        }else 
      left = center +1 ;
     }
 return ans ; 

}
__global__  void solution_sizx_seven (ll* edges ,ll* four_cliques , ll* three_cliques ,ll  m ,ll number_of_four_cli  ,ll three_size ,ll*  update , ll k  ) 
{
    
     ll index = blockDim.x * blockIdx.x + threadIdx.x ;
    if(index < number_of_four_cli)
    {
        update[index] = 0 ; 
        if(k==6)
        {
            //4 and 3 







               ll starting_element = find_starting_three(three_cliques , 0 , three_size -1 ,  four_cliques[index*4+3])  ;  // last element  
                if(starting_element == -1 )
                {
                 return ; 
                }
          
          for( ; starting_element< three_size && three_cliques[starting_element*3] ==  four_cliques[index*4+3] ; starting_element++ )
          {
                int counted = 1 ; 
                for(int j =0 ; j < 3 ; j++)
                {
                    for(int k = 0; k <4 ; k++)
                    {
                        //four clique .. two_clique 
                     if(j!=0 && k!=3  )
                        if(check_edge(edges,0,m-1, three_cliques[starting_element*3+j ] ,four_cliques[index*4+k])== 0    )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
             update[index] += counted ; 

          }
        }else 
     {
               ll starting_element = find_starting_four(four_cliques , 0 , number_of_four_cli -1 ,  four_cliques[index*4+3])  ;  // last element  
                if(starting_element == -1 )
                {
                 return ; 
                }
          
          for( ; starting_element< number_of_four_cli && four_cliques[starting_element*4] ==  four_cliques[index*4+3] ; starting_element++ )
          {
                int counted = 1 ; 
                for(int j =0 ; j < 4 ; j++)
                {
                    for(int k = 0; k <4 ; k++)
                    {
                        //four .. three 
                     if(j!=0 && k!=3  )
                        if(check_edge(edges,0,m-1, four_cliques[starting_element*4+j ] ,four_cliques[index*4+k])== 0    )
                        {
                            counted =0 ; 
                            break ;    
                        }
                    }
                }
             update[index] += counted ; 

          }
         
     }
     
    }
}



//------------------------------------------------------------------

void  take_input( vector<ll> & edges , vector<ll> & two_clique , ll m )
{
  vector<pair<ll,ll>> unpacked_two_cli ; 
  for(ll i=0;i<m;i++)
  {
      
      ll a , b ; indata>>a>>b ;
      if(a>b)
      swap(a,b) ; 
      unpacked_two_cli.push_back({a,b}) ; 
  }
 
  sort(unpacked_two_cli.begin() , unpacked_two_cli.end()) ;
  vector<pair<ll,ll>> unpacked_edges = unpacked_two_cli; 
  for(auto i :  unpacked_two_cli )
  {
      unpacked_edges.push_back({i.second , i.first}) ; 
  }
  sort(unpacked_edges.begin(),  unpacked_edges.end()); 
 for(ll i=0;i<m;i++)
 {
     
 two_clique[i*2] = unpacked_two_cli[i].first ;
  
 two_clique[i*2+1] = unpacked_two_cli[i].second ;
 }


 for(ll i=0;i<2*m;i++)
 {
     
 edges[i*2] = unpacked_edges[i].first ;
  
 edges[i*2+1] = unpacked_edges[i].second ;
 }




}



//--------------------------------------------------------------------------------
ll solution(string file_name,  ll k )
{   
    
    indata.open(file_name); // opens the file
    if(!indata) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
    }
    
 //   freopen("com-youtube.ungraph.txt","r",stdin); //as20000102
    
    ll n = 1 ;
    ll  m ; indata>> m ; 

    vector<ll> edges(4*m) , two_cliques(2*m) ;
    take_input(edges ,  two_cliques , m ) ; 
    for(auto  i : two_cliques)
    n = max(i , n ) ; 
      if(k==2)
      return m ; 
 

    n++  ;
    m*=2 ; 
 
if(k==1)
return n-1 ; 
 if(k==2)
 return m/2 ; 
 else 
    {
      
      /*--------------------------------------------------------------Copy the edges and two cliques----------------------------------------------*/
      ll *device_edges  , *device_updates , *device_two_cliques ;
  

      hipMalloc( &device_edges ,  sizeof(ll)*m*2)  ;
      hipMalloc( &device_two_cliques , sizeof(ll)*(m) )  ;

      hipMemcpy( device_edges , edges.data() , sizeof(ll)*m*2 , hipMemcpyHostToDevice )  ;   
      hipMemcpy(  device_two_cliques , two_cliques.data() , sizeof(ll)*m , hipMemcpyHostToDevice )  ;   

      /*------------------------------------------------------------------------------------------------------------------------------------------*/
      






      /*---------------------------------------------------------------Count number of three cliques----------------------------------------------*/
      vector<ll> start_3_cli (m/2)  ;
      ll * device_start_3_cli ; 
      hipMalloc( &device_start_3_cli , sizeof(ll)*m/2) ; 
      
      
      // Invoke kernel
      int threadsPerBlock = 100 ;
      ll blocksPerGrid = ( (m/2 )+ threadsPerBlock - 1) / threadsPerBlock;
      count_three_cliques<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_two_cliques,   n , m , device_start_3_cli  );
       

      hipMemcpy(start_3_cli.data() , device_start_3_cli, sizeof(ll)*(m/2) , hipMemcpyDeviceToHost);
      ll current_size =0 ; 
     
      for(ll i= 0;i<m/2 ;i++)
      {    
         ll temp =start_3_cli[i]  ; 
         start_3_cli[i] = current_size ; 
        current_size += temp ; 
      }
      /*----------------------------------------------------------------------------------------------------------------------------------------*/
     
     
      if(k==3)
      {
          hipFree(device_start_3_cli) ; 
          hipFree(device_edges) ; 
          hipFree(device_updates) ;
          hipFree(device_two_cliques); 
          return current_size ; 
      }


     /*-------------------------------------------------Copy all the three cliques-------------------------------------------------------------*/

      if(current_size ==0 )
        return 0 ; 
      ll* three_cliques  ; 
      hipMalloc( &three_cliques , sizeof(ll)*current_size*3) ;
       
      hipMemcpy(device_start_3_cli ,start_3_cli.data()  , sizeof(ll)*(m/2) , hipMemcpyHostToDevice )  ;   
    
       make_three_cliques<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_two_cliques,   n , m , device_start_3_cli , three_cliques   );

    /*-----------------------------------------------------------------------------------------------------------------------------------------*/
     
    ll sol_ans =0 ;
 

    if(k <6)
    {
     /*----------------------------------------------------------------Find the solutions-------------------------------------------------------*/
        threadsPerBlock = 1024 ;
       blocksPerGrid = ( (current_size )+ threadsPerBlock - 1) / threadsPerBlock;
      
      
           vector<ll>updates(current_size) ;
         hipMalloc( &device_updates , sizeof(ll)*(current_size) )  ;

     
        solution_greater<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_two_cliques, three_cliques,    n , m , k ,current_size , device_updates );
         hipMemcpy(updates.data() , device_updates, sizeof(ll)*(current_size) , hipMemcpyDeviceToHost);

      for(auto i :  updates) 
        sol_ans +=  i ;
     
    /*------------------------------------------------------------------------------------------------------------------------------------------*/
        
    }else 
     {
         
         /*----------------------------------------------------------------Find the solutions-------------------------------------------------------*/
        threadsPerBlock = 1024 ;
       blocksPerGrid = ( (current_size )+ threadsPerBlock - 1) / threadsPerBlock;
      
        ll *four_clique_size ; 
        hipMalloc( &four_clique_size , sizeof(ll)*(current_size) )  ;

     
        solution_greater<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_two_cliques, three_cliques,    n , m , 4 ,current_size , four_clique_size ); // We only need 4 cliques 
        vector<ll> four_cli_si(current_size ) ; 
        hipMemcpy(four_cli_si.data() , four_clique_size, sizeof(ll)*(current_size) , hipMemcpyDeviceToHost);
  
        ll number_of_four_cli =  0 ;
        for(ll i= 0;i<current_size;i++)
        {
            ll temp = four_cli_si[i] ; 
            four_cli_si[i] = number_of_four_cli ; 
            number_of_four_cli += temp ;
        }
      
        hipMemcpy(  four_clique_size,four_cli_si.data() ,  sizeof(ll)*(current_size) , hipMemcpyHostToDevice) ;
        
        //sol_ans = number_of_four_cli ; 
            /*-------------------------------------------------------------Now make the four clique----------------------------------------------*/
              ll* four_cliques ; 
              hipMalloc(&four_cliques, sizeof(ll)*4*number_of_four_cli ) ; 
              make_four_clique<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_two_cliques, three_cliques,    n , m , k ,current_size , four_clique_size , four_cliques );
            /*------------------------------------------------------------------------------------------------------------------------------------*/
      


            /*----------------------------------------------------------Now ask the values of 6 and 7 cliques-------------------------------------*/
            ll *device_updates ; 
              blocksPerGrid = ( (number_of_four_cli )+ threadsPerBlock - 1) / threadsPerBlock;
              hipMalloc(&device_updates , sizeof(ll)*number_of_four_cli ) ;
          
                        vector<ll> arsd(number_of_four_cli) ; 
                       solution_sizx_seven<<<blocksPerGrid,threadsPerBlock>>> (device_edges , four_cliques , three_cliques , m , number_of_four_cli  , current_size,  device_updates , k ) ; 
                    
                      hipMemcpy(arsd.data() , device_updates, sizeof(ll)*number_of_four_cli , hipMemcpyDeviceToHost);  

                      
                          for(ll i =0;i<  number_of_four_cli ;i++)
                              sol_ans += arsd[i] ;                   
      
                      

            /*------------------------------------------------------------------------------------------------------------------------------------*/



      
    /*------------------------------------------------------------------------------------------------------------------------------------------*/
     


     }
     

      hipFree(device_updates);
      hipFree(device_edges);
      hipFree(device_two_cliques);
      hipFree(three_cliques);
      hipFree(device_start_3_cli);
     
      return sol_ans ; 
     
    }
}



int main()
{

      string file_name  ; cin>>file_name ; 
    
    ll k ; cin>>k  ;
    time_t x1, x2 ;
    x1 = time(NULL);
    
    string sd =""  ;
    ll ans = solution(file_name, k) ;
    x2 = time(NULL) ; 
    cout<<"\n"<<x2-x1<<" " << k<<" "<<ans  ; 

 

}