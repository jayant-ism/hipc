//////////////////////////////////////////New code////////////////////////////////////


#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>
#include<set>
#include<fstream>
#include <chrono>


using namespace std::chrono;
using std::ifstream;
ifstream indata ; 

#define ll long long int 


using namespace std;
/*----------------------------------------------------------------------------*/

__device__ bool check_edge(ll* edges  , ll left ,ll right , ll ff , ll ss )
{
 
    while(left<=right)
    {
        ll center= (right-left)/2 + left ; 
   //  printf("%d %d %d %d \n" , edges[center*2] , edges[center*2+1] , ff , ss  ) ; 
        if(edges[center*2]== ff && edges[center*2+1] == ss)
        return 1 ;  //Found 

        if(edges[center*2] > ff )
        {
            right = center -1 ; 
        }else if(edges[center*2] < ff )
        left = center+1 ; 
        else if (edges[center*2+1] < ss)
        left = center +1 ; 
        else 
        right = center -1 ; 
     }
    return 0 ; 
}

/*----------------------------------------------------------------------------*/

__global__ void make_count (ll* device_edges , ll* device_last_ele  ,ll current_cli_size ,ll m ,ll* device_count )
{
    
      ll index = blockDim.x * blockIdx.x + threadIdx.x ;
    if(index < current_cli_size )
    {
        device_count[index] =0  ; 
        for(ll i = index +1 ; i<current_cli_size && device_last_ele[index*2] == device_last_ele[i*2] ; i++)
        {
            if(check_edge(device_edges  , 0 , m-1 , device_last_ele[index*2+1] , device_last_ele[i*2+1]))
            device_count[index]++ ; //Since it is sorted we need not worry about anything other  
        }
    }
}
    
/*----------------------------------------------------------------------------*/

__global__ void  make_next(ll* device_edges ,ll* device_last_ele  ,ll current_cli_size ,ll m ,ll* device_count ,ll* device_new_last )
{
    
      ll index = blockDim.x * blockIdx.x + threadIdx.x ;
    if(index < current_cli_size )
    {
        ll starting_ele = device_count[index] ;
        for(ll i = index+ 1  ; i<current_cli_size && device_last_ele[index*2] == device_last_ele[i*2] ; i++)
        {
            if(check_edge(device_edges  , 0 , m-1 , device_last_ele[index*2+1] , device_last_ele[i*2+1]))
            {
                device_new_last[2*starting_ele] = index ; 
                device_new_last[1+2*starting_ele] = device_last_ele[i*2+1] ; 
                starting_ele++ ; 
            }
            
        }
    }
}
       
/*----------------------------------------------------------------------------*/







void  make_two_cli(ll m  , vector<ll>& host_edges , vector<ll>& host_two_cli  )
{ 
    set<pair<ll,ll>> tem ;  
    for(ll i =0;i<m;i++)
    {
        ll a,b ; indata>>a>>b ; 
        tem.insert({a,b})  ;    //Considering the directed edges 
        tem.insert({b,a})  ;    //Considering the directed edges 
    
    }
    for(auto i : tem)
    {
        
        /*
        //Directed
      if(i.first < i.second && (tem.find({i.second , i.first }) != tem.end()) )
         {host_two_cli.push_back(i.first) ;  host_two_cli.push_back(i.second) ;} 
        */
      host_edges.push_back(i.first);host_edges.push_back(i.second); //It will be automatically sorted 
     
      if(i.first < i.second )
         {host_two_cli.push_back(i.first) ;  host_two_cli.push_back(i.second) ; }
       
     
    }
 //So I have updated  the value of both 
}

ll find_kcliq(ll k)
{
    ll  m ;indata>>m ;
    ll n = 1 ;
    /// take the input 
    vector<ll> host_edges; 
    vector<ll> host_cli;
    make_two_cli(m , host_edges , host_cli) ;
    ll current_cli_size = host_cli.size()/2 ; //Num of cliques  
    for(auto i : host_cli)
      n = max(i,n) ; 
    if(k==1)
    return n ; 
 
    if(k==2)
    return current_cli_size ; 
    n++ ; 
    
    m =  host_edges.size()/2  ; 
    
    ll cli = 2 ;
    ll *device_last_ele  , *device_edges ; 
    
    
    hipMalloc( &device_edges , sizeof(ll)*m*2) ;
    hipMalloc( &device_last_ele , sizeof(ll)*current_cli_size*2) ;

    hipMemcpy(device_edges ,host_edges.data()  , sizeof(ll)*(m*2) , hipMemcpyHostToDevice )  ;   
    hipMemcpy(device_last_ele ,host_cli.data()  , sizeof(ll)*(current_cli_size*2) , hipMemcpyHostToDevice )  ;   
    
    while(cli < k )
    {
        //We need to count the number of next nodes 

        ll *device_count ; 
        vector<ll> count(current_cli_size) ; 
        hipMalloc( &device_count  , sizeof(ll)*current_cli_size) ;
        

        int threadsPerBlock = 100 ;
        ll blocksPerGrid = ( (current_cli_size)+ threadsPerBlock - 1) / threadsPerBlock;
        make_count<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_last_ele  , current_cli_size , m , device_count );
        hipMemcpy(count.data() , device_count, sizeof(ll)*(current_cli_size) , hipMemcpyDeviceToHost);

        ll new_cli_size =0  ;
        for(ll i =0;i<current_cli_size ; i++)
        {
            ll temp =  count[i] ; 
            count[i] = new_cli_size ; 
         new_cli_size += temp ; 
        }
      
        hipMemcpy(device_count,count.data()   , sizeof(ll)*current_cli_size, hipMemcpyHostToDevice )  ;
  
     if(new_cli_size==0)
     {
         current_cli_size = 0 ; 
      break ; 
     }
     
     if(1+cli < k   )
     {
         //Updete the value
        ll  *device_new_last ; 
        hipMalloc( &device_new_last  , sizeof(ll)*new_cli_size*2) ;

        make_next<<<blocksPerGrid, threadsPerBlock>>> (device_edges , device_last_ele  , current_cli_size , m , device_count  , device_new_last );
        hipFree(device_last_ele) ; 
        device_last_ele = device_new_last ; 
       
     }
     

     cli++ ; 
     current_cli_size = new_cli_size ; 
 
    }
 
    hipFree(device_edges) ; 
    hipFree(device_last_ele) ;
    return  current_cli_size ; 
 
    


}
int main()
{
    
    ll k ;
    //cin>> k ;
    string file_name   ;
    cin>>file_name >>k ; 
    
    indata.open(file_name); // opens the file
    if(!indata) { // file couldn't be opened
      cerr << "Error: file could not be opened" << endl;
      exit(1);
    }
  milliseconds ms = duration_cast< milliseconds >(
    system_clock::now().time_since_epoch()
);
 
    ll ms1 =std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    ll ans  =  find_kcliq(k) ;
    ll ms2 = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    
    cout<< ans  <<"\nExecution Time: "<< ms2 - ms1 <<"ms" ;

}
